#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include "mpi.h"

#define _USE_MATH_DEFINES

#define XMIN 0.0
#define XMAX 1.0
#define YMIN 0.0
#define YMAX 1.0

#define MAX_PSZ 10
#define TSCALE 1.0
#define VSQR 0.1

void init(double *u, double *pebbles, int n);
void evolve(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t);
void evolve9pt(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t);
int tpdt(double *t, double dt, double end_time);
void print_heatmap(const char *filename, double *u, int n, double h);
void init_pebbles(double *p, int pn, int n);

void run_cpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time);

extern void run_gpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time, int nthreads);

int my_rank,m_size;
int tag=50;
MPI_Status status;


int main(int argc, char *argv[])
{

  if(argc != 5)
  {
    printf("Usage: %s npoints npebs time_finish nthreads \n",argv[0]);
    return 0;
  }

  int     npoints   = atoi(argv[1]);
  int     npebs     = atoi(argv[2]);
  double  end_time  = (double)atof(argv[3]);
  int     nthreads  = atoi(argv[4]);
  int 	  narea	    = npoints * npoints;
	
  if(npoints%4!=0){
   printf("Enter number of points which are divisible by 4!");
   return 0;
  }  
  int number_of_nodes;

  MPI_Init(&argc,&argv);

  MPI_Comm_rank(MPI_COMM_WORLD,&my_rank);
  MPI_Comm_size(MPI_COMM_WORLD,&number_of_nodes);
  printf("Number of nodes %d\n",number_of_nodes);
  /*if(number_of_nodes!=4){
  	printf("Error: Use 4 nodes!\n");
	MPI_Finalize();
	return 0;
  }
*/
/* m_size is memory required on every node, on the 
* boundry nodes you just have n extra values and other 
* node you will have 2n extra values received from the peer node.
*/  
  m_size = (narea/4);

  if(my_rank==0 || my_rank==3){
	m_size = m_size + npoints;
  } else {
  	m_size = m_size + (2*npoints);
  }
  
  double *u_i0, *u_i1;
  double *u_cpu, *u_gpu, *pebs;
  double h;

  double elapsed_cpu, elapsed_gpu;
  struct timeval cpu_start, cpu_end, gpu_start, gpu_end;
  
  u_i0 = (double*)malloc(sizeof(double) * m_size);
  u_i1 = (double*)malloc(sizeof(double) * m_size);
  if(my_rank==0){
  	pebs = (double*)malloc(sizeof(double) * npoints*npoints);
  } else {
	pebs = (double*)malloc(sizeof(double)*m_size);
  }
  u_cpu = (double*)malloc(sizeof(double) * m_size);
  u_gpu = (double*)malloc(sizeof(double) * m_size);

  printf("Running %s with (%d x %d) grid, until %f, with %d threads\n", argv[0], npoints, npoints, end_time, nthreads);

  h = (XMAX - XMIN)/npoints;
  //Send the pebbles value calculate from the first node
  if(my_rank==0){
  	init_pebbles(pebs, npebs, npoints);
  	MPI_Send(pebs+npoints,((npoints*npoints)/4)+(2*npoints),MPI_DOUBLE,1,tag,MPI_COMM_WORLD);
  	MPI_Send(pebs+(3*npoints),((npoints*npoints)/4)+(2*npoints),MPI_DOUBLE,2,tag,MPI_COMM_WORLD);
  	MPI_Send(pebs+(5*npoints),((npoints*npoints)/4)+(npoints),MPI_DOUBLE,3,tag,MPI_COMM_WORLD);
	
  }else{
	//recv the pebbels 
	MPI_Recv(pebs,m_size,MPI_DOUBLE,0,tag,MPI_COMM_WORLD,&status);
  }

  init(u_i0, pebs, npoints);
  init(u_i1, pebs, npoints);
  
   char heat[20];
  sprintf(heat,"lake_i_%d.dat",my_rank);
  print_heatmap(heat, u_i0, npoints, h);

  gettimeofday(&cpu_start, NULL);
  run_cpu(u_cpu, u_i0, u_i1, pebs, npoints, h, end_time);
  gettimeofday(&cpu_end, NULL);

  elapsed_cpu = ((cpu_end.tv_sec + cpu_end.tv_usec * 1e-6)-(
                  cpu_start.tv_sec + cpu_start.tv_usec * 1e-6));
  printf("CPU took %f seconds\n", elapsed_cpu);

  /*gettimeofday(&gpu_start, NULL);
  run_gpu(u_gpu, u_i0, u_i1, pebs, npoints, h, end_time, nthreads);
  gettimeofday(&gpu_end, NULL);
  elapsed_gpu = ((gpu_end.tv_sec + gpu_end.tv_usec * 1e-6)-(
                  gpu_start.tv_sec + gpu_start.tv_usec * 1e-6));
  printf("GPU took %f seconds\n", elapsed_gpu);
*/
  char filename[20];
  sprintf(filename,"lake_f_%d.dat",my_rank); 
  print_heatmap(filename, u_cpu, npoints, h);
  //sprintf(filename,"lakegpu_f_%d.dat",my_rank); 
  //print_heatmap("lakegpu_f.dat", u_gpu, npoints, h);
  

  free(u_i0);
  free(u_i1);
  free(pebs);
  free(u_cpu);
  free(u_gpu);
  MPI_Finalize();
  return 1;
}

void run_cpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time)
{
  double *un, *uc, *uo;
  double t, dt;

  un = (double*)malloc(sizeof(double) * m_size);
  uc = (double*)malloc(sizeof(double) * m_size);
  uo = (double*)malloc(sizeof(double) * m_size);

  memcpy(uo, u0, sizeof(double) * m_size);
  memcpy(uc, u1, sizeof(double) * m_size);

  t = 0.;
  dt = h / 2.;

  while(1)
  {
	if(my_rank==0){
		MPI_Send(uo+(m_size-(2*n)),n,MPI_DOUBLE,1,tag,MPI_COMM_WORLD);
		MPI_Send(uc+(m_size-(2*n)),n,MPI_DOUBLE,1,tag,MPI_COMM_WORLD);
		MPI_Recv(uo+(m_size-n),n,MPI_DOUBLE,1,tag,MPI_COMM_WORLD,&status);
		MPI_Recv(uc+(m_size-n),n,MPI_DOUBLE,1,tag,MPI_COMM_WORLD,&status);
	} else if(my_rank==1){
		MPI_Recv(uo,n,MPI_DOUBLE,0,tag,MPI_COMM_WORLD,&status);
		MPI_Recv(uc,n,MPI_DOUBLE,0,tag,MPI_COMM_WORLD,&status);
		MPI_Send(uo+n,n,MPI_DOUBLE,0,tag,MPI_COMM_WORLD);
		MPI_Send(uc+n,n,MPI_DOUBLE,0,tag,MPI_COMM_WORLD);
		MPI_Send(uo+(m_size-(2*n)),n,MPI_DOUBLE,2,tag,MPI_COMM_WORLD);
		MPI_Send(uc+(m_size-(2*n)),n,MPI_DOUBLE,2,tag,MPI_COMM_WORLD);
		MPI_Recv(uo+(m_size-n),n,MPI_DOUBLE,2,tag,MPI_COMM_WORLD,&status);
		MPI_Recv(uc+(m_size-n),n,MPI_DOUBLE,2,tag,MPI_COMM_WORLD,&status);
	} else if(my_rank==2){
		MPI_Recv(uo,n,MPI_DOUBLE,1,tag,MPI_COMM_WORLD,&status);
		MPI_Recv(uc,n,MPI_DOUBLE,1,tag,MPI_COMM_WORLD,&status);
		MPI_Send(uo+n,n,MPI_DOUBLE,1,tag,MPI_COMM_WORLD);
		MPI_Send(uc+n,n,MPI_DOUBLE,1,tag,MPI_COMM_WORLD);
		MPI_Send(uo+(m_size-(2*n)),n,MPI_DOUBLE,3,tag,MPI_COMM_WORLD);
		MPI_Send(uc+(m_size-(2*n)),n,MPI_DOUBLE,3,tag,MPI_COMM_WORLD);
		MPI_Recv(uo+(m_size-n),n,MPI_DOUBLE,3,tag,MPI_COMM_WORLD,&status);
		MPI_Recv(uc+(m_size-n),n,MPI_DOUBLE,3,tag,MPI_COMM_WORLD,&status);
	}else{
		MPI_Recv(uo,n,MPI_DOUBLE,2,tag,MPI_COMM_WORLD,&status);
		MPI_Recv(uc,n,MPI_DOUBLE,2,tag,MPI_COMM_WORLD,&status);
		MPI_Send(uo+n,n,MPI_DOUBLE,2,tag,MPI_COMM_WORLD);
		MPI_Send(uc+n,n,MPI_DOUBLE,2,tag,MPI_COMM_WORLD);
		
	}
    //evolve(un, uc, uo, pebbles, n, h, dt, t);
    evolve9pt(un, uc, uo, pebbles, n, h, dt, t);
	
    memcpy(uo, uc, sizeof(double) * m_size);
    memcpy(uc, un, sizeof(double) * m_size);

    if(!tpdt(&t,dt,end_time)) break;
  }
  
  memcpy(u, un, sizeof(double) * m_size);
}

void init_pebbles(double *p, int pn, int n)
{
  int i, j, k, idx;
  int sz;

  srand( time(NULL) );
  memset(p, 0, sizeof(double) * n * n);

  for( k = 0; k < pn ; k++ )
  {
    i = rand() % (n - 4) + 2;
    j = rand() % (n - 4) + 2;
    sz = rand() % MAX_PSZ;
    idx = j + i * n;
    p[idx] = (double) sz;
  }
}

double f(double p, double t)
{
  return -expf(-TSCALE * t) * p;
}

int tpdt(double *t, double dt, double tf)
{
  if((*t) + dt > tf) return 0;
  (*t) = (*t) + dt;
  return 1;
}

void init(double *u, double *pebbles, int n)
{
  int i, j, idx;

  for(i = 0; i < n/4; i++)
  {
    for(j = 0; j < n ; j++)
    {
      idx = j + i * n;
      u[idx] = f(pebbles[idx], 0.0);
    }
  }
}

void evolve(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t)
{
  int i, j, idx;

  for( i = 0; i < n; i++)
  {
    for( j = 0; j < n; j++)
    {
      idx = j + i * n;

      if( i == 0 || i == n - 1 || j == 0 || j == n - 1)
      {
        un[idx] = 0.;
      }
      else
      {
        un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + 
                    uc[idx + n] + uc[idx - n] - 4 * uc[idx])/(h * h) + f(pebbles[idx],t));
      }
    }
  }
}
void evolve9pt(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t)
{
  int i, j, idx,k;
  if(my_rank==0 ||my_rank==3){
  	k=(n/4)+1;
  }else{
  	k=(n/4)+2;
  }

  for( i = 0; i < k; i++)
  {
    for( j = 0; j < n; j++)
    {
      idx = j + i * n;

      if( i == 0 || i == n - 1 || j == 0 || j == n - 1)
      {
        un[idx] = 0.;
      }
      else
      {
        un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + 
                    uc[idx + n] + uc[idx - n] +0.25*(uc[idx-n+1]+uc[idx+n-1]+uc[idx-n-1]+uc[idx+n+1])  - 5 * uc[idx])/(h * h) + f(pebbles[idx],t));
      }
    }
  }
}

void print_heatmap(const char *filename, double *u, int n, double h)
{
  int i, j, idx,k,start=0;

  FILE *fp = fopen(filename, "w");  
  k=n/4;
  if(my_rank!=0){
	start = 1;
  }
  
  for( i = start; i < k; i++ )
  {
    for( j = 0; j < n; j++ )
    {
      idx = j + i * n;
      fprintf(fp, "%f %f %f\n", i*h, j*h, u[idx]);
    }
  }
  
  fclose(fp);
} 
